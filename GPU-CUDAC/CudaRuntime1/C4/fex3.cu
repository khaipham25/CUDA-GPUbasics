
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void kernel1() {
	printf("Kernel1\n");
}
__global__ void kernel2() {
	printf("Kernel2\n");
}
int main() {
	kernel1 <<<1, 1 >>> ();
	printf("CPU here\n");
	kernel2 <<<1, 1 >>> ();
	hipDeviceSynchronize();
	printf("here");
	return 0;
}