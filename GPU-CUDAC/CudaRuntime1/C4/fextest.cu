#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
__global__ void kernel() {
	printf("GPU\n");
}
int main() {
	kernel << <2, 5 >> > ();
	hipDeviceSynchronize();
	return 0;
}