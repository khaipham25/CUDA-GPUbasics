﻿#include <stdio.h>
#include "hip/hip_runtime.h"

__device__ void Device1() {
	printf("Device11\n");
}
__device__ void Device2() {
	printf("Device22\n");
	//Func() trong này không thể dùng
}
void Func() {
	//Device1(); cũng không thể sử dụng
}
int main() {
	Func();
	//Device2(); không thể sử dụng như này mà chỉ trong __global__ mới được
	hipDeviceSynchronize();
	return 0;
}