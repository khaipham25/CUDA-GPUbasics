#include "hip/hip_runtime.h"


#include <stdio.h>
__global__ void kernel() {
	int temp=0;
	temp = threadIdx.x;
	printf("blockId %d threadId %d = %d\n", blockIdx.x, threadIdx.x, temp);

}
int main() {
	kernel <<<5, 5 >>> ();
	hipDeviceSynchronize();
	return 0;
}