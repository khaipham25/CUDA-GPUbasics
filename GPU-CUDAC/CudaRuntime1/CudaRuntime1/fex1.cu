#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__device__ void Device1() {
	printf("Device1\n");
}
__device__ void Device2() {
	printf("Device2\n");
}
__global__ void kernel() {
	Device1();
	Device2();
}
__device__ void Device3() {
	//khong sd kernel<<<1,1>>>
}
void Func_in_host() {
	kernel << <1, 1 >> > ();
	hipDeviceSynchronize();//đợi GPU hoàn thành trước khi tiếp tục ,CPU có thể thực hiện công việc khác
}
int main() {
	Func_in_host();
}