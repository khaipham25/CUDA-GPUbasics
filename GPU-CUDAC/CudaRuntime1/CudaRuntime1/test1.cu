#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__device__ void Device1() {
	printf("Device1\n");
}
__device__ void Device22() {
	printf("Device3\n");
}
__global__ void kernel() {
	Device1();
	Device2();
}
void Func_in_host() {
	kernel <<<1, 1 >>> ();
	hipDeviceSynchronize();//đợi GPU hoàn thành trước khi tiếp tục ,CPU có thể thực hiện công việc khác
}
int main() {
	Func_in_host();
}