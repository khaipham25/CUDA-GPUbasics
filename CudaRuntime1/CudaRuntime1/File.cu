#include "hip/hip_runtime.h"
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

// H�m kernel tr�n GPU
__global__ void hello()
{
    printf("Hello from Thread %d in Block %d\n", threadIdx.x, blockIdx.x);
}

int main()
{
    hello < <<1, 1 >> > ();

    // ??ng b? h�a GPU v� CPU
    hipDeviceSynchronize();

    return 0;
}
